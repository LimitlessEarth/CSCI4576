#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include "timer.h"

#define SOFTENING 1e-3f
#define BLOCK_SIZE 256

typedef struct { float4 *pos; float3 *vel; } Particle;

void print_usage() {
    printf("Usage: -d image dimensions\n-t time step\n-i number of iterations\n-w writing on off\n-p number of particles\n");
}

void initialize_particles(float *data, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < 4; j++) {
            if (j == 3) {
                data[i * 4 + j] = 10.0f; //1.5f * (rand() / (float)RAND_MAX + 10.0);
            } else {
                data[i * 4 + j] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
            }
        }
    }
}

__global__
void body_force(float4 *p, float3 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float3 a = {0.0f, 0.0f, 0.0f};

        for (int tile = 0; tile < gridDim.x; tile++) {
            __shared__ float4 spos[BLOCK_SIZE];
            float4 tpos = p[tile * blockDim.x + threadIdx.x];
            spos[threadIdx.x] = make_float4(tpos.x, tpos.y, tpos.z, tpos.w);
            __syncthreads();

            #pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j++) {
                float3 r;
                r.x = spos[j].x - p[i].x;
                r.y = spos[j].y - p[i].y;
                r.z = spos[j].z - p[i].z;
                
                float dist_sqr = r.x*r.x + r.y*r.y + r.z*r.z + SOFTENING;
                float inv_dist = rsqrtf(dist_sqr);
                float inv_dist3 = inv_dist * inv_dist * inv_dist;

                float ac = spos[j].w * inv_dist3;

                a.x += r.x * ac; 
                a.y += r.y * ac; 
                a.z += r.z * ac;
            }
            __syncthreads();
        }

        v[i].x += a.x * dt; 
        v[i].y += a.y * dt; 
        v[i].z += a.z * dt;
    }
}

int main(int argc, char* argv[]) {
    
    int                 num_part            = 1000000;
    int                 bytes               = num_part * sizeof(float4) + num_part * sizeof(float3);
    int                 num_iter            = 10;    // simulation iterations   
    float               dt                  = 0.0001f;    // time step
    int                 img_dim             = 1500;
    int                 option              = -1;
    double              writing_time        = 0;
    bool                writing             = false;
    int                 img_len;
    int                 nBlocks, frame, i;
    float               *buf, *d_buf;
    double              total_frame_time, avg_time, comp_time;
    int                 loc, x, y, a;
    char                frame_name[47];
    char                *out_buffer;

    // Parse commandline
    while ((option = getopt(argc, argv, "d:t:i:wp:")) != -1) {        
        switch (option) {
             case 'd' : 
                 img_dim = atoi(optarg);
                 break;
             case 't' : 
                 dt = atoi(optarg) * 0.0001;
                 break;
             case 'i' : 
                 num_iter = atoi(optarg);
                 break;
             case 'w' :
                 writing = true;
                 break;
             case 'p' :
                 num_part = atoi(optarg);
                 break;
             default:
                 print_usage(); 
                 exit(1);
        }
    }
    
    img_len = img_dim * img_dim;
    
    buf = (float*) malloc(bytes);
    Particle Host_Particle = { (float4*)buf, ((float3*)buf) + num_part };

    initialize_particles(buf, num_part); // Init pos / vel data / mass
    
    out_buffer = (char *) calloc(img_dim * img_dim, sizeof(char));

    hipMalloc(&d_buf, bytes);
    Particle Device_Particle = { (float4*) d_buf, ((float3*)d_buf) + num_part };

    nBlocks = (num_part + BLOCK_SIZE - 1) / BLOCK_SIZE;
    total_frame_time = 0.0;

        //for (i = 0 ; i < num_part; i++) {
            //printf("X: %f\tY: %f\tZ: %f\tMASS: %f\n", Host_Particle.pos[i].x, Host_Particle.pos[i].y, Host_Particle.pos[i].z, Host_Particle.pos[i].w);
        //}

    for (frame = 1; frame <= num_iter; frame++) {
        StartTimer();

        hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
        body_force<<<nBlocks, BLOCK_SIZE>>>(Device_Particle.pos, Device_Particle.vel, dt, num_part);
        hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

        for (i = 0 ; i < num_part; i++) { // integrate position
            Host_Particle.pos[i].x += Host_Particle.vel[i].x * dt;
            Host_Particle.pos[i].y += Host_Particle.vel[i].y * dt;
            Host_Particle.pos[i].z += Host_Particle.vel[i].z * dt;
            //printf("X: %f\tY: %f\tZ: %f\tMASS: %f\n", Host_Particle.pos[i].x, Host_Particle.pos[i].y, Host_Particle.pos[i].z, Host_Particle.pos[i].w);
        }

        comp_time = GetTimer() / 1000.0;
        if (frame > 1) { // First iter is warm up
            total_frame_time += comp_time; 
        }
        
        if (writing) {
            StartTimer();
 
            // write out pgm
            for (a = 0; a < num_part; a++) {
                x = (int) (Host_Particle.pos[a].x * 100.0) + (img_dim / 2);
                y = (int) (Host_Particle.pos[a].y * 100.0) + (img_dim / 2);
    
                if (x < 0 || x > img_dim)
                    continue;
    
                loc = x + (img_dim * y);
                //printf("%d\n", loc);
                if (loc >= 0 && loc < img_len) {        
                    out_buffer[loc] = 255;
                }
    
            }
   
            sprintf(frame_name, "img/%d.pgm", frame);
            FILE *file = fopen(frame_name, "w");
            fprintf(file, "P5\n");
            fprintf(file, "%d %d\n", img_dim, img_dim);
            fprintf(file, "%d\n", 255);
            fwrite(out_buffer, sizeof(char), img_len, file);
            fclose(file);

            for (a = 0; a < num_part; a++) {
                x = (int) (Host_Particle.pos[a].x * 100.0) + (img_dim / 2);
                y = (int) (Host_Particle.pos[a].y * 100.0) + (img_dim / 2);
    
                if (x < 0 || x > img_dim)
                    continue;
    
                loc = x + (img_dim * y);
                if (loc >= 0 && loc < img_len) {        
                    out_buffer[loc] = 0;
                }
            }
        
            writing_time = GetTimer() / 1000.0;
        }
        
        printf("Iteration %d:\t%.10f seconds\t%f seconds\n", frame, comp_time, writing_time);
        
    }
    
    avg_time = total_frame_time / (double) (num_iter-1); 

    printf("Total computation time was: %f\t\tAverage frame time was: %f\t\tAverage Particle interations per second were: %f\n", total_frame_time, avg_time, (double) (num_part * num_part) / avg_time);
    
    free(out_buffer);
    free(buf);
    hipFree(d_buf);
}
