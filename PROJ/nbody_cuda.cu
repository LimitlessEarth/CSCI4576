#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include "timer.h"

#define SOFTENING 1e-9f
#define BLOCK_SIZE 256

typedef struct { float4 *pos, *vel; } Particle;

void print_usage() {
    printf("Usage: -d image dimensions\n-t time step\n-i number of iterations\n-w writing on off\n-p number of particles\n");
}

void initialize_particles(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__
void body_force(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float ax = 0.0f; float ay = 0.0f; float az = 0.0f;

        for (int tile = 0; tile < gridDim.x; tile++) {
            __shared__ float3 spos[BLOCK_SIZE];
            float4 tpos = p[tile * blockDim.x + threadIdx.x];
            spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
            __syncthreads();

            #pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j++) {
                float dx = spos[j].x - p[i].x;
                float dy = spos[j].y - p[i].y;
                float dz = spos[j].z - p[i].z;
                float dist_sqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                float inv_dist = rsqrtf(dist_sqr);
                float inv_dist3 = inv_dist * inv_dist * inv_dist;

                ax += dx * inv_dist3; 
                ay += dy * inv_dist3; 
                az += dz * inv_dist3;
            }
            __syncthreads();
        }

        v[i].x += ax * dt; 
        v[i].y += ay * dt; 
        v[i].z += az * dt;
    }
}

int main(int argc, char* argv[]) {
    
    int                 num_part            = 1000000;
    int                 bytes               = 2 * num_part * sizeof(float4);
    int                 num_iter            = 10;    // simulation iterations   
    float               dt                  = 0.0001f;    // time step
    int                 img_dim             = 1500;
    int                 option              = -1;
    double              writing_time        = 0;
    bool                writing             = false;
    int                 img_len;
    int                 nBlocks, frame, i;
    float               *buf, *d_buf;
    double              total_frame_time, avg_time, comp_time;
    int                 loc, x, y, a;
    char                frame_name[47];
    char                *out_buffer;

    // Parse commandline
    while ((option = getopt(argc, argv, "d:t:i:wp:")) != -1) {        
        switch (option) {
             case 'd' : 
                 img_dim = atoi(optarg);
                 break;
             case 't' : 
                 dt = atoi(optarg) * 0.0001;
                 break;
             case 'i' : 
                 num_iter = atoi(optarg);
                 break;
             case 'w' :
                 writing = true;
                 break;
             case 'p' :
                 num_part = atoi(optarg);
                 break;
             default:
                 print_usage(); 
                 exit(1);
        }
    }
    
    img_len = img_dim * img_dim;
    
    buf = (float*) malloc(bytes);
    Particle Host_Particle = { (float4*)buf, ((float4*)buf) + num_part };

    initialize_particles(buf, 8 *num_part); // Init pos / vel data
    
    out_buffer = (char *) calloc(img_dim * img_dim, sizeof(char));

    hipMalloc(&d_buf, bytes);
    Particle Device_Particle = { (float4*) d_buf, ((float4*)d_buf) + num_part };

    nBlocks = (num_part + BLOCK_SIZE - 1) / BLOCK_SIZE;
    total_frame_time = 0.0; 

    for (frame = 1; frame <= num_iter; frame++) {
        StartTimer();

        hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
        body_force<<<nBlocks, BLOCK_SIZE>>>(Device_Particle.pos, Device_Particle.vel, dt, num_part);
        hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

        for (i = 0 ; i < num_part; i++) { // integrate position
            Host_Particle.pos[i].x += Host_Particle.vel[i].x * dt;
            Host_Particle.pos[i].y += Host_Particle.vel[i].y * dt;
            Host_Particle.pos[i].z += Host_Particle.vel[i].z * dt;
        }

        comp_time = GetTimer() / 1000.0;
        if (frame > 1) { // First iter is warm up
            total_frame_time += comp_time; 
        }
        
        if (writing) {
            StartTimer();
 
            // write out pgm
            for (a = 0; a < num_part; a++) {
                x = (int) (Host_Particle.pos[a].x * 100.0) + (img_dim / 2);
                y = (int) (Host_Particle.pos[a].y * 100.0) + (img_dim / 2);
    
                loc = x + (img_dim * y);
                //printf("%d\n", loc);
                if (loc >= 0 && loc < img_len) {        
                    out_buffer[loc] = 255;
                }
    
            }
   
            sprintf(frame_name, "img/%d.pgm", frame);
            FILE *file = fopen(frame_name, "w");
            fprintf(file, "P5\n");
            fprintf(file, "%d %d\n", img_dim, img_dim);
            fprintf(file, "%d\n", 255);
            fwrite(out_buffer, sizeof(char), img_len, file);
            fclose(file);

            for (a = 0; a < num_part; a++) {
                x = (int) (Host_Particle.pos[a].x * 100.0) + (img_dim / 2);
                y = (int) (Host_Particle.pos[a].y * 100.0) + (img_dim / 2);
    
                loc = x + (img_dim * y);
                if (loc >= 0 && loc < img_len) {        
                    out_buffer[loc] = 0;
                }
            }
        
            writing_time = GetTimer() / 1000.0;
        }
        
        printf("Iteration %d:\t%.10f seconds\t%f seconds\n", frame, comp_time, writing_time);
        
    }
    
    avg_time = total_frame_time / (double) (num_iter-1); 

    printf("Total computation time was: %f\t\tAverage frame time was: %f\t\tAverage Particle interations per second were: %f\n", total_frame_time, avg_time, (double) (num_part * num_part) / avg_time);
    
    free(out_buffer);
    free(buf);
    hipFree(d_buf);
}
