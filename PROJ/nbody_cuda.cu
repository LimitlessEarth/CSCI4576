#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f


typedef struct { float4 *pos, *vel; } Particle;

void initialize_particles(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__
void body_force(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

        for (int tile = 0; tile < gridDim.x; tile++) {
            __shared__ float3 spos[BLOCK_SIZE];
            float4 tpos = p[tile * blockDim.x + threadIdx.x];
            spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
            __syncthreads();

            #pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j++) {
                float dx = spos[j].x - p[i].x;
                float dy = spos[j].y - p[i].y;
                float dz = spos[j].z - p[i].z;
                float dist_sqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                float inv_dist = rsqrtf(dist_sqr);
                float inv_dist3 = inv_dist * inv_dist * inv_dist;

                Fx += dx * inv_dist3; 
                Fy += dy * inv_dist3; 
                Fz += dz * inv_dist3;
            }
            __syncthreads();
        }

        v[i].x += Fx * dt; 
        v[i].y += Fy * dt; 
        v[i].z += Fz * dt;
    }
}

int main(const int argc, const char** argv) {
    
    int                 num_part            = 500000;
    int                 bytes               = 2 * num_part * sizeof(float4);
    const int           num_iter            = 100;    // simulation iterations   
    const float         dt                  = 0.01f;    // time step
    int                 nBlocks, frame, i;
    float               *buf, *d_buf;
    double              total_time, avg_time;
    
    buf = (float*) malloc(bytes);
    Particle Host_Particle = { (float4*)buf, ((float4*)buf) + num_part };

    initialize_particles(buf, 8 *num_part); // Init pos / vel data

    hipMalloc(&d_buf, bytes);
    Particle Device_Particle = { (float4*) d_buf, ((float4*)d_buf) + num_part };

    nBlocks = (num_part + BLOCK_SIZE - 1) / BLOCK_SIZE;
    total_time = 0.0; 

    for (frame = 1; frame <= num_iter; frame++) {
        StartTimer();

        hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
        body_force<<<nBlocks, BLOCK_SIZE>>>(Device_Particle.pos, Device_Particle.vel, dt, num_part);
        hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

        for (i = 0 ; i < num_part; i++) { // integrate position
            Host_Particle.pos[i].x += Host_Particle.vel[i].x * dt;
            Host_Particle.pos[i].y += Host_Particle.vel[i].y * dt;
            Host_Particle.pos[i].z += Host_Particle.vel[i].z * dt;
        }

        const double time_elapsed = GetTimer() / 1000.0;
        if (frame > 1) { // First iter is warm up
            total_time += time_elapsed; 
        }
        printf("Iteration %d: %.20f seconds\n", iter, time_elapsed);
    }
    avg_time = total_time / (double) (num_iter-1); 

    printf("%d, %0.3f\n", num_part, 1e-9 * num_part * num_part / avg_time);
    //printf("Average rate for iterations 2 through %d: %.3f +- %.3f steps per second.\n", num_iter, rate);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", num_part, 1e-9 * num_part * num_part / avg_time);
    
    free(buf);
    hipFree(d_buf);
}
