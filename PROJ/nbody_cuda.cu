#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f


typedef struct { float4 *pos, *vel; } Particle;

void initialize_particles(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        printf("%f\n", data[i]);
    }
}

__global__
void body_force(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float ax = 0.0f; float ay = 0.0f; float az = 0.0f;

        for (int tile = 0; tile < gridDim.x; tile++) {
            __shared__ float3 spos[BLOCK_SIZE];
            float4 tpos = p[tile * blockDim.x + threadIdx.x];
            spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
            __syncthreads();

            #pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j++) {
                float dx = spos[j].x - p[i].x;
                float dy = spos[j].y - p[i].y;
                float dz = spos[j].z - p[i].z;
                float dist_sqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                float inv_dist = rsqrtf(dist_sqr);
                float inv_dist3 = inv_dist * inv_dist * inv_dist;

                ax += dx * inv_dist3; 
                ay += dy * inv_dist3; 
                az += dz * inv_dist3;
            }
            __syncthreads();
        }

        v[i].x += ax * dt; 
        v[i].y += ay * dt; 
        v[i].z += az * dt;
    }
}

int main(const int argc, const char** argv) {
    
    int                 num_part            = 500000;
    int                 bytes               = 2 * num_part * sizeof(float4);
    const int           num_iter            = 100;    // simulation iterations   
    const float         dt                  = 0.01f;    // time step
    int                 img_dim             = 1500;
    int                 nBlocks, frame, i;
    float               *buf, *d_buf;
    double              total_time, avg_time;
    int                 loc, x, y, a;
    char                frame_name[47];
    char                *out_buffer;
    
    buf = (float*) malloc(bytes);
    Particle Host_Particle = { (float4*)buf, ((float4*)buf) + num_part };

    initialize_particles(buf, 8 *num_part); // Init pos / vel data
    
    out_buffer = (char *) calloc(img_dim * img_dim, sizeof(char));

    hipMalloc(&d_buf, bytes);
    Particle Device_Particle = { (float4*) d_buf, ((float4*)d_buf) + num_part };

    nBlocks = (num_part + BLOCK_SIZE - 1) / BLOCK_SIZE;
    total_time = 0.0; 

    for (frame = 1; frame <= num_iter; frame++) {
        StartTimer();

        hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
        body_force<<<nBlocks, BLOCK_SIZE>>>(Device_Particle.pos, Device_Particle.vel, dt, num_part);
        hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

        for (i = 0 ; i < num_part; i++) { // integrate position
            Host_Particle.pos[i].x += Host_Particle.vel[i].x * dt;
            Host_Particle.pos[i].y += Host_Particle.vel[i].y * dt;
            Host_Particle.pos[i].z += Host_Particle.vel[i].z * dt;
        }

        const double time_elapsed = GetTimer() / 1000.0;
        if (frame > 1) { // First iter is warm up
            total_time += time_elapsed; 
        }
        printf("Iteration %d: %.20f seconds\n", frame, time_elapsed);
        
        // write out pgm
        for (i = 0; i < num_part; i++) {
        
            printf("%f", Host_Particle.pos[i].x);
    
            /*for (a = 0; a < num_part; a++) {        
                x = (Host_Particle.pos[i].x / DOMAIN_SIZE) * img_dim;        
                y = (Host_Particle.pos[i].y / DOMAIN_SIZE) * img_dim;
        
                loc = x + (img_dim * y);
                if (loc >= 0 && loc < img_len) {        
                    out_buffer[loc] = 255;
                }
        
            }
       
            sprintf(frame_name, "img/%d.pgm", n);
            FILE *file = fopen(frame_name, "w");
            fprintf(file, "P5\n");
            fprintf(file, "%d %d\n", img_dim, img_dim);
            fprintf(file, "%d\n", 255);
            fwrite(out_buffer, sizeof(char), img_len, file);
            fclose(file);
    
            for (a = 0; a < num_part; a++) {
                x = (Particles_a[a].pos[X] / DOMAIN_SIZE) * img_dim;        
                y = (Particles_a[a].pos[Y] / DOMAIN_SIZE) * img_dim;
                
                loc = x + (img_dim * y);
                if (loc >= 0 && loc < img_len) {        
                    out_buffer[loc] = 0;
                }    
            }*/
        }
    }
    
    avg_time = total_time / (double) (num_iter-1); 

    printf("%d, %0.3f\n", num_part, 1e-9 * num_part * num_part / avg_time);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", num_part, 1e-9 * num_part * num_part / avg_time);
    
    free(out_buffer);
    free(buf);
    hipFree(d_buf);
}
